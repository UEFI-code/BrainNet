
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<unistd.h>
#define N 16
#define PoolEnergy 1.0
#define CellEnergy 100
#define MaxOut 10.0
#define RecoverySpeed 5.0
#define tiredRecoveryRate 0.6
float iobuf[N] = {0.0};
float *iobuf_gpu = 0;
float weights[N * (N-1)] = {0.0};
float *weights_gpu = 0;
float cellConsume[N] = {0.0};
float *cellConsume_gpu = 0;
__global__ void Neural(float *iobuf_gpu, float *weights_gpu, float *cellConsume_gpu) 
{
	float x = 0.0;
	for(int i=0;i<N-1;i++)
        {
		if(i<threadIdx.x)
			x += iobuf_gpu[i] * weights_gpu[threadIdx.x*(N-1)+i];
		else
			x += iobuf_gpu[i+1] * weights_gpu[threadIdx.x*(N-1)+i];
        }
	if(x>MaxOut) x = (x/(x+1))*MaxOut;
	if(x<0-MaxOut) x = (x/(1-x))*MaxOut;
	if(CellEnergy - cellConsume_gpu[threadIdx.x] > 0)
	{
		x = x * (CellEnergy - cellConsume_gpu[threadIdx.x]) / CellEnergy;
		if(x>0.0)
			cellConsume_gpu[threadIdx.x] -= (RecoverySpeed - x) * PoolEnergy;
		else
			cellConsume_gpu[threadIdx.x] -= (RecoverySpeed + x) * PoolEnergy;
		if(cellConsume_gpu[threadIdx.x] < 0) cellConsume_gpu[threadIdx.x] = 0;
	}
	else
	{
		cellConsume_gpu[threadIdx.x] = cellConsume_gpu[threadIdx.x] * tiredRecoveryRate;
		x = 0.0;
	}
	iobuf_gpu[threadIdx.x] = x;
}

int main()
{
	srand((unsigned)time(NULL));
	for(int i=0;i<(N * (N-1));i++)
	{
		weights[i] = (rand()%200)/100.0;
		if((rand()%2) == 0)
			weights[i] = 0 - weights[i];
	}
	iobuf[0] = (rand()%200)/100.0;
	hipMalloc(&iobuf_gpu, sizeof(float)*N);
	hipMalloc(&weights_gpu, sizeof(float)*N*(N-1));
	hipMalloc(&cellConsume_gpu, sizeof(float)*N);
	hipMemcpy(iobuf_gpu, iobuf, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemcpy(weights_gpu, weights, sizeof(float)*N*(N-1), hipMemcpyHostToDevice);
	hipMemcpy(cellConsume_gpu, cellConsume, sizeof(float)*N, hipMemcpyHostToDevice);
	while(1)
	{
		Neural<<<1,N>>>(iobuf_gpu,weights_gpu,cellConsume_gpu);
		hipMemcpy(iobuf, iobuf_gpu, sizeof(float)*N, hipMemcpyDeviceToHost);
        	hipMemcpy(cellConsume, cellConsume_gpu, sizeof(float)*N, hipMemcpyDeviceToHost);
        	for(int i=0;i<N;i++)
                	printf("%.2f ",iobuf[i]);
		printf("\n");
		sleep(1);
	}
		
}
