
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<unistd.h>
#define N 16
#define PoolEnergy 1.0
#define CellEnergy 25.0
#define MaxOut 10.0
#define RecoverySpeed 3.0
#define tiredRecoveryRate 0.6
float iobuf[N] = {0.0};
float *iobuf_gpu = 0;
float weights[N * (N-1)] = {0.0};
float *weights_gpu = 0;
float cellConsume[N] = {0.0};
float *cellConsume_gpu = 0;
__global__ void Neural(float *iobuf_gpu, float *weights_gpu, float *cellConsume_gpu, float exciteNum) 
{
	float x = 0.0;
	bool flag = true;
	for(int i=0;i<N-1;i++)
        {
		if(i<threadIdx.x)
			x += iobuf_gpu[i] * weights_gpu[threadIdx.x*(N-1)+i];
		else
			x += iobuf_gpu[i+1] * weights_gpu[threadIdx.x*(N-1)+i];
        }
	if(threadIdx.x < exciteNum)
	{
		flag = true;
		if(x>0.0)
		{
			if(x>MaxOut) x = (x/(x+1))*MaxOut;
		}
		else x = 0.0;
	}
	else
	{
		flag = false;
		x = 0 - x;
		if(x<0.0)
		{
			if(x<0-MaxOut) x = (x/(1-x))*MaxOut;
		}
		else x = 0.0;
	}
	if(CellEnergy - cellConsume_gpu[threadIdx.x] > 0)
	{
		x = x * (CellEnergy - cellConsume_gpu[threadIdx.x]) / CellEnergy;
		if(flag)
			cellConsume_gpu[threadIdx.x] -= RecoverySpeed * PoolEnergy - x;
		else
			cellConsume_gpu[threadIdx.x] -= RecoverySpeed * PoolEnergy + x;
		if(cellConsume_gpu[threadIdx.x] < 0) cellConsume_gpu[threadIdx.x] = 0;
	}
	else
	{
		cellConsume_gpu[threadIdx.x] = cellConsume_gpu[threadIdx.x] * tiredRecoveryRate;
		x = 0.0;
	}
	iobuf_gpu[threadIdx.x] = x;
}

int main()
{
	srand((unsigned)time(NULL));
	for(int i=0;i<(N * (N-1));i++)
	{
		weights[i] = (rand()%100)/100.0;
	}
	hipMalloc(&iobuf_gpu, sizeof(float)*N);
	hipMalloc(&weights_gpu, sizeof(float)*N*(N-1));
	hipMalloc(&cellConsume_gpu, sizeof(float)*N);
	hipMemcpy(weights_gpu, weights, sizeof(float)*N*(N-1), hipMemcpyHostToDevice);
	hipMemcpy(cellConsume_gpu, cellConsume, sizeof(float)*N, hipMemcpyHostToDevice);
	while(1)
	{
		iobuf[0] = (rand()%1000)/100.0;
                hipMemcpy(iobuf_gpu, iobuf, sizeof(float)*N, hipMemcpyHostToDevice);
		Neural<<<1,N>>>(iobuf_gpu,weights_gpu,cellConsume_gpu,N);
		hipMemcpy(iobuf, iobuf_gpu, sizeof(float)*N, hipMemcpyDeviceToHost);
        	hipMemcpy(cellConsume, cellConsume_gpu, sizeof(float)*N, hipMemcpyDeviceToHost);
        	for(int i=0;i<N;i++)
                	printf("%.2f ",iobuf[i]);
		printf("\n");
		sleep(1);
	}
		
}
