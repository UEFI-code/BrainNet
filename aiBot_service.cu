
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<unistd.h>
#include <sys/shm.h>
#define N 256
#define PoolEnergy 1.0
#define CellEnergy 25.0
#define MaxOut 10.0
#define RecoverySpeed 3.0
#define tiredRecoveryRate 0.6
float *iobuf = 0;
float *iobuf_gpu = 0;
float *weights = 0;
float *weights_gpu = 0;
float *cellConsume = 0;
float *cellConsume_gpu = 0;
__global__ void Neural(float *iobuf_gpu, float *weights_gpu, float *cellConsume_gpu, float exciteNum) 
{
	float x = 0.0;
	bool flag = true;
	for(int i=0;i<N-1;i++)
        {
		if(i<threadIdx.x)
			x += iobuf_gpu[i] * weights_gpu[threadIdx.x*(N-1)+i];
		else
			x += iobuf_gpu[i+1] * weights_gpu[threadIdx.x*(N-1)+i];
        }
	if(threadIdx.x < exciteNum)
	{
		flag = true;
		if(x>0.0)
		{
			if(x>MaxOut) x = (x/(x+1))*MaxOut;
		}
		else x = 0.0;
	}
	else
	{
		flag = false;
		x = 0 - x;
		if(x<0.0)
		{
			if(x<0-MaxOut) x = (x/(1-x))*MaxOut;
		}
		else x = 0.0;
	}
	if(CellEnergy - cellConsume_gpu[threadIdx.x] > 0)
	{
		x = x * (CellEnergy - cellConsume_gpu[threadIdx.x]) / CellEnergy;
		if(flag)
			cellConsume_gpu[threadIdx.x] -= (RecoverySpeed - x) * PoolEnergy;
		else
			cellConsume_gpu[threadIdx.x] -= (RecoverySpeed + x) * PoolEnergy;
		if(cellConsume_gpu[threadIdx.x] < 0) cellConsume_gpu[threadIdx.x] = 0;
	}
	else
	{
		cellConsume_gpu[threadIdx.x] = cellConsume_gpu[threadIdx.x] * tiredRecoveryRate;
		x = 0.0;
	}
	iobuf_gpu[threadIdx.x] = x;
}

int main()
{
	srand((unsigned)time(NULL));
	int shmid = shmget((key_t)2333, sizeof(float)*N, 0777|IPC_CREAT);
        iobuf = (float *)shmat(shmid, 0, 0);
	shmid = shmget((key_t)2334, sizeof(float)*N*(N-1), 0777|IPC_CREAT);
	weights = (float *)shmat(shmid, 0, 0);
	shmid = shmget((key_t)2335, sizeof(float)*N, 0777|IPC_CREAT);
        cellConsume = (float *)shmat(shmid, 0, 0);
	for(int i=0;i<N;i++)
	{
		iobuf[i] = 0;
		cellConsume[i] = 0;
	}
	for(int i=0;i<(N * (N-1));i++)
		weights[i] = (rand()%100)/100.0;
	hipMalloc(&iobuf_gpu, sizeof(float)*N);
	hipMalloc(&weights_gpu, sizeof(float)*N*(N-1));
	hipMalloc(&cellConsume_gpu, sizeof(float)*N);
	while(1)
	{
		hipMemcpy(weights_gpu, weights, sizeof(float)*N*(N-1), hipMemcpyHostToDevice);
        	hipMemcpy(cellConsume_gpu, cellConsume, sizeof(float)*N, hipMemcpyHostToDevice);
                hipMemcpy(iobuf_gpu, iobuf, sizeof(float)*N, hipMemcpyHostToDevice);
		Neural<<<1,N>>>(iobuf_gpu,weights_gpu,cellConsume_gpu,N-4);
		hipMemcpy(iobuf, iobuf_gpu, sizeof(float)*N, hipMemcpyDeviceToHost);
        	hipMemcpy(cellConsume, cellConsume_gpu, sizeof(float)*N, hipMemcpyDeviceToHost);
        	
		hipMemcpy(weights, weights_gpu, sizeof(float)*N, hipMemcpyDeviceToHost);
	}
		
}
